#include <stdio.h>  // for printf
#include <stdlib.h> // for malloc
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

#define UNKNOWN_KEYSIZE 11
#define MEMORY_ALLOCATION_PROBLEM 33

// Implementation: S-Box

unsigned char sbox[256] = {
    // 0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,  // 0
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,  // 1
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,  // 2
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,  // 3
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,  // 4
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,  // 5
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,  // 6
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,  // 7
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,  // 8
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,  // 9
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,  // A
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,  // B
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,  // C
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,  // D
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,  // E
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16}; // F

// Implementation: Rcon
unsigned char Rcon[255] = {

    0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8,
    0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3,
    0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f,
    0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d,
    0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab,
    0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d,
    0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25,
    0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01,
    0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d,
    0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa,
    0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a,
    0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02,
    0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a,
    0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef,
    0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94,
    0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04,
    0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f,
    0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5,
    0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33,
    0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb};


enum keySize
{
    SIZE_16 = 16,
    SIZE_24 = 24,
    SIZE_32 = 32
};

/* Rijndael's key schedule rotate operation
 * rotate the word eight bits to the left
 *
 * rotate(1d2c3a4f) = 2c3a4f1d
 *
 * word is an char array of size 4 (32 bit)
 */
void rotate(unsigned char *word)
{
    unsigned char c;
    int i;

    c = word[0];
    for (i = 0; i < 3; i++)
        word[i] = word[i + 1];
    word[3] = c;
}

void core(unsigned char *word, int iteration)
{
    int i;

    /* rotate the 32-bit word 8 bits to the left */
    rotate(word);

    /* apply S-Box substitution on all 4 parts of the 32-bit word */
    for (i = 0; i < 4; ++i)
    {
        word[i] = sbox[word[i]];
    }

    /* XOR the output of the rcon operation with i to the first part (leftmost) only */
    word[0] = word[0] ^ Rcon[iteration];
}

/* Rijndael's key expansion
 * expands an 128,192,256 key into an 176,208,240 bytes key
 *
 * expandedKey is a pointer to an char array of large enough size
 * key is a pointer to a non-expanded key
 */

void expandKey(unsigned char *expandedKey,
               unsigned char *key,
               enum keySize size,
               size_t expandedKeySize)
{
    /* current expanded keySize, in bytes */
    int currentSize = 0;
    int rconIteration = 1;
    int i;
    unsigned char t[4] = {0}; // temporary 4-byte variable

    /* set the 16,24,32 bytes of the expanded key to the input key */
    for (i = 0; i < size; i++)
        expandedKey[i] = key[i];
    currentSize += size;

    while (currentSize < expandedKeySize)
    {
        /* assign the previous 4 bytes to the temporary value t */
        for (i = 0; i < 4; i++)
        {
            t[i] = expandedKey[(currentSize - 4) + i];
        }

        /* every 16,24,32 bytes we apply the core schedule to t
         * and increment rconIteration afterwards
         */
        if (currentSize % size == 0)
        {
            core(t, rconIteration++);
        }

        /* For 256-bit keys, we add an extra sbox to the calculation */
        if (size == SIZE_32 && ((currentSize % size) == 16))
        {
            for (i = 0; i < 4; i++)
                t[i] = sbox[t[i]];
        }

        /* We XOR t with the four-byte block 16,24,32 bytes before the new expanded key.
         * This becomes the next four bytes in the expanded key.
         */
        for (i = 0; i < 4; i++)
        {
            expandedKey[currentSize] = expandedKey[currentSize - size] ^ t[i];
            currentSize++;
        }
    }
}

__device__ void subBytes(unsigned char *state, unsigned char *sbox)
{
    int i;
    /* substitute all the values from the state with the value in the SBox
     * using the state value as index for the SBox
     */
    for (i = 0; i < 16; i++)
        state[i] = sbox[state[i]];
}

__device__ void shiftRow(unsigned char *state, unsigned char nbr)
{
    int i, j;
    unsigned char tmp;
    /* each iteration shifts the row to the left by 1 */
    for (i = 0; i < nbr; i++)
    {
        tmp = state[0];
        for (j = 0; j < 3; j++)
            state[j] = state[j + 1];
        state[3] = tmp;
    }
}

__device__ void shiftRows(unsigned char *state)
{
    int i;
    /* iterate over the 4 rows and call shiftRow() with that row */
    for (i = 0; i < 4; i++)
        shiftRow(state + i * 4, i);
}

__device__ void addRoundKey(unsigned char *state, unsigned char *roundKey)
{
    int i;
    for (i = 0; i < 16; i++)
        state[i] = state[i] ^ roundKey[i];
}

__device__ unsigned char galois_multiplication(unsigned char a, unsigned char b)
{
    unsigned char p = 0;
    unsigned char counter;
    unsigned char hi_bit_set;
    for (counter = 0; counter < 8; counter++)
    {
        if ((b & 1) == 1)
            p ^= a;
        hi_bit_set = (a & 0x80);
        a <<= 1;
        if (hi_bit_set == 0x80)
            a ^= 0x1b;
        b >>= 1;
    }
    return p;
}

__device__ void mixColumn(unsigned char *column)
{
    unsigned char cpy[4];
    int i;
    for (i = 0; i < 4; i++)
    {
        cpy[i] = column[i];
    }
    column[0] = galois_multiplication(cpy[0], 2) ^
                galois_multiplication(cpy[3], 1) ^
                galois_multiplication(cpy[2], 1) ^
                galois_multiplication(cpy[1], 3);

    column[1] = galois_multiplication(cpy[1], 2) ^
                galois_multiplication(cpy[0], 1) ^
                galois_multiplication(cpy[3], 1) ^
                galois_multiplication(cpy[2], 3);

    column[2] = galois_multiplication(cpy[2], 2) ^
                galois_multiplication(cpy[1], 1) ^
                galois_multiplication(cpy[0], 1) ^
                galois_multiplication(cpy[3], 3);

    column[3] = galois_multiplication(cpy[3], 2) ^
                galois_multiplication(cpy[2], 1) ^
                galois_multiplication(cpy[1], 1) ^
                galois_multiplication(cpy[0], 3);
}

__device__ void mixColumns(unsigned char *state)
{
    int i, j;
    unsigned char column[4];

    /* iterate over the 4 columns */
    for (i = 0; i < 4; i++)
    {
        /* construct one column by iterating over the 4 rows */
        for (j = 0; j < 4; j++)
        {
            column[j] = state[(j * 4) + i];
        }

        /* apply the mixColumn on one column */
        mixColumn(column);

        /* put the values back into the state */
        for (j = 0; j < 4; j++)
        {
            state[(j * 4) + i] = column[j];
        }
    }
}

__device__ void aes_round(unsigned char *state, unsigned char *roundKey, unsigned char *sbox)
{
    subBytes(state, sbox);
    shiftRows(state);
    mixColumns(state);
    addRoundKey(state, roundKey);
}

__device__ void createRoundKey(unsigned char *expandedKey, unsigned char *roundKey)
{
    int i, j;
    /* iterate over the columns */
    for (i = 0; i < 4; i++)
    {
        /* iterate over the rows */
        for (j = 0; j < 4; j++)
            roundKey[(i + (j * 4))] = expandedKey[(i * 4) + j];
    }
}

__global__ void aes_main(unsigned char *input, unsigned char *output, unsigned char *expandedKey, int nbrRounds, unsigned char *sbox, int msg_length)
{
    int id = threadIdx.x;
    int blockId = blockIdx.x;

    __shared__ unsigned char d_sbox[256];
    __shared__ unsigned char d_expandedKey[176];

    if(id < 256) {
        d_sbox[id] = sbox[id];
    }

    if(id < 176) {
        d_expandedKey[id] = expandedKey[id];
    }

    __syncthreads();

    if ((id + 16) <= msg_length) {
        int i;

        unsigned char state[16];
        for (i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
                state[(i + (j * 4))] = input[(blockId * msg_length) + (16 * id) + ((i * 4) + j)];
        }

        i = 0;
        unsigned char roundKey[16];

        createRoundKey(d_expandedKey, roundKey);
        addRoundKey(state, roundKey);

        for (i = 1; i < nbrRounds; i++)
        {
            createRoundKey(d_expandedKey + 16 * i, roundKey);
            aes_round(state, roundKey, d_sbox);
        }

        createRoundKey(d_expandedKey + 16 * nbrRounds, roundKey);
        subBytes(state, d_sbox);
        shiftRows(state);
        addRoundKey(state, roundKey);

        for (i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
                output[(blockId * msg_length) + (16 * id) + ((i * 4) + j)] = state[(i + (j * 4))];
        }
    }
}

char aes_encrypt(unsigned char *input,
                 unsigned char *output,
                 unsigned char *key,
                 enum keySize size,
                 int num_msgs,
                 int msg_length)
{

    unsigned char *d_sbox;
    unsigned char *d_input;
    unsigned char *d_output;
    unsigned char *d_expandedKey;

    hipMalloc((void **) &d_sbox, 256);
    hipMemcpy(d_sbox, sbox, 256, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_input, num_msgs*msg_length);
    hipMalloc((void **) &d_output, num_msgs*msg_length);
    hipMemcpy(d_input, input, num_msgs*msg_length, hipMemcpyHostToDevice);

    /* the expanded keySize */
    int expandedKeySize;

    /* the number of rounds */
    int nbrRounds;

    /* the expanded key */
    unsigned char *expandedKey;

    /* set the number of rounds */
    switch (size)
    {
    case SIZE_16:
        nbrRounds = 10;
        break;
    case SIZE_24:
        nbrRounds = 12;
        break;
    case SIZE_32:
        nbrRounds = 14;
        break;
    default:
        return UNKNOWN_KEYSIZE;
        break;
    }

    expandedKeySize = (16 * (nbrRounds + 1));
    if ((expandedKey = (unsigned char*)(malloc(expandedKeySize * sizeof(char)))) == NULL)
    {
        return MEMORY_ALLOCATION_PROBLEM;
    }

    /* Set the block values, for the block:
     * a0,0 a0,1 a0,2 a0,3
     * a1,0 a1,1 a1,2 a1,3
     * a2,0 a2,1 a2,2 a2,3
     * a3,0 a3,1 a3,2 a3,3
     * the mapping order is a0,0 a1,0 a2,0 a3,0 a0,1 a1,1 ... a2,3 a3,3
     */

    /* expand the key into an 176, 208, 240 bytes key */
    expandKey(expandedKey, key, size, expandedKeySize);
    hipMalloc((void**) &d_expandedKey, 176);
    hipMemcpy(d_expandedKey, expandedKey, 176, hipMemcpyHostToDevice);

    /* encrypt the block using the expandedKey */
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    aes_main<<<num_msgs, 1024>>>(d_input, d_output, d_expandedKey, nbrRounds, d_sbox, msg_length);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Time taken by encrypt on GPU:  %10.6f microseconds \n", time*1000);

    hipEventRecord(start, 0);
    hipMemcpy(output, d_output, num_msgs*msg_length, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Time taken by CUDA memcpy:  %10.6f microseconds \n", time*1000);

    return 0;
}

int main(int argc, char *argv[])
{
    // the cipher key
    unsigned char key[16] = {'k', 'k', 'k', 'k', 'e', 'e', 'e', 'e', 'y', 'y', 'y', 'y', '.', '.', '.', '.'};

    // the plaintext
    int msg_length = 64;
    int num_msgs = 64;
    unsigned char plaintext[num_msgs*msg_length];
    for(int i = 0; i < num_msgs; i++) {
        for(int j = 0; j < (msg_length); j++)
            plaintext[i * msg_length + j] = '0' + (j % 10);
    }

    // the ciphertext
    unsigned char ciphertext[num_msgs*msg_length];

    // the decrypted text
    // unsigned char decryptedtext[msg_length];

    int i;

    printf("\nCipher Key:\n");
    for (i = 0; i < 16; i++)
    {
        printf("%2.2x%c", key[i], ((i + 1) % 16) ? ' ' : '\n');
    }

    printf("\nPlaintext:\n");
    for (i = 0; i < msg_length; i++)
    {
        printf("%2.2x%c", plaintext[i], ((i + 1) % 16) ? ' ' : '\n');
    }

    // AES Encryption
    aes_encrypt(plaintext, ciphertext, key, SIZE_16, num_msgs, msg_length);

    printf("\nCiphertext:\n");
    for (i = msg_length; i < 2*msg_length; i++)
    {
        printf("%2.2x%c", ciphertext[i], ((i + 1) % 16) ? ' ' : '\n');
    }

    return 0;
}